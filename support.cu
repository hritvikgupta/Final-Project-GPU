#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

#include "support.h"
//Error handling using functions of the CUDA runtime API
void cudaCheckError() {                                                              
  hipError_t e=hipGetLastError();                                                   
  if(e!=hipSuccess) {                                                                
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));   
      hipDeviceReset();                                                              
      exit(EXIT_FAILURE);                                                             
  }                                                                                   
}

//This macro checks malloc() and hipMalloc() return values
void Check_Allocation_Return_Value(double *a){   
  if(a==NULL) {                           
  printf("Allocation Error\n");           
  hipDeviceReset();                      
  exit(EXIT_FAILURE);                     
  }                                       
}
void verify(double *hA, double *hB, double *hC, unsigned int m) {

    printf("Check results...\n");
    int i,j,k;
    int N = m;
    double res; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            res=0;
            for(k=0;k<N;k++){
                res+=hA[i*N+k]*hB[k*N+j];
            }
            
           //printf("%8.3f ",res);
           if(res != hC[i*N+j]){
                printf("NOT OK i:%d, j:%d\n",i,j);
                printf("true value:%f - computed value:%f\n\n",res,hC[i*N+j]);
           }
        }
        //printf("\n");
    }
  printf("TEST PASSED");

}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

